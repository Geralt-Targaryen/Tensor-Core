#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "include/utils.hpp"
#include "include/resnet.hpp"
using namespace std;


int main(int argc, char *argv[]){

    int h=224, w=224, c=3, N=5000, bs = argc>1? atoi(argv[1]):8;
    bool forward_with_float=false;
    int h_=1, w_=5000, c_=1;
    int input_size = h * w * c, output_size = h_ * w_ * c_;
    double time_argmax=0;
    time_t tic, toc, tic1, toc1;
    
    resnet18 model(forward_with_float);
    model.load_weights("input/param.bin");

    // load images
    vector<float> input(input_size * N);
    vector<int> label(N), label_(N);
    cout<<"Loading data..."<<endl;
    load_binary(input, "input/data.bin", input_size * N);
    load_binary(label, "input/label.bin", N);
    printf("Loaded data of shape (%d, %d, %d, %d)\n", (int)input.size()/(h*w*c), c, h, w);

    float *d_input, *d_output;
    hipMalloc((void**)(&d_input), sizeof(float) * input_size * bs);

    cout<<"Starting inference with batch size "<<bs<<"..."<<endl;
    tic = time(NULL);
    for(int i=0; i<N; i+=bs){
        int bs_ = min(bs, N-i);
        hipMemcpy(d_input, input.data() + i * input_size, sizeof(float) * input_size * bs_, hipMemcpyHostToDevice);

        d_output = model.forward(d_input, h, w, bs_);
        tic1 = clock();
        int *prediction = argmax(d_output, bs_, 1000);
        toc1 = clock();
        time_argmax += double(toc1-tic1)/1000000;
        hipMemcpy(label_.data() + i, prediction, sizeof(int) * bs_, hipMemcpyDeviceToHost);
        hipFree(prediction);
        hipFree(d_output);
        
    }
    toc = time(NULL);
    model.compute_time();

    hipFree(d_input);

    cout<<"time - conv2d: "<<model.time_conv2d<<"s\n";
    cout<<"\ttime - im2col: "<<model.time_im2col<<"s\n";
    cout<<"\ttime - gemm: "<<model.time_gemm<<"s\n";
    cout<<"time - batchnorm2d: "<<model.time_batchnorm2d<<"s\n";
    cout<<"time - relu: "<<model.time_relu<<"s\n";
    cout<<"time - maxpool2d: "<<model.time_maxpool2d<<"s\n";
    cout<<"time - adaptiveavgpool2d: "<<model.time_adaptiveavgpool2d<<"s\n";
    cout<<"time - linear: "<<model.time_linear<<"s"<<endl;
    cout<<"time - argmax: "<<time_argmax<<"s\n"<<endl;
    
    cout<<"Inference time: "<<difftime(toc, tic)<<"s"<<endl;
    cout<<"Test acc: "<<accuracy_score(label.data(), label_.data(), N)<<endl;
}
